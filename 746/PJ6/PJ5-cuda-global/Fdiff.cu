#include "hip/hip_runtime.h"
/*************************

   File: Fdiff_CUDA.c
   Compile: nvcc Fdiff_CUDA.cu Fdutils_CUDA.cu -O3 -o FdGPU -lm
   Use: ./Fd [input file] [tile size]  | qsub -d `pwd` -q fermi.q
        ./Fd [input file] [tile size]  | qsub -d `pwd` -q g92.q

   Performs 4 nearest neighbor updates on 2-D grid
   Input file format:

   # cycles
   width of grid (including boundary)
   # initial data points

   3 integers per data point: i and j indices, data


*************************/

#include <stdio.h>
#include <stdlib.h>
#include "Fdiff.h"
#include "hip/hip_runtime.h"

int main(int arg, char **argv) {
  int width;
  int numCycles;
  int i, j, n;
  float *u0, *u1, *tptr;
  double inTemp;
  int cycle = 0;
  int numInit;

  // value of second command line argument
  //int TILE_SIZE = atoi(argv[2]);

  //printf("Tile: %d\n", TILE_SIZE);
  //faceprintf("success.\n");
  FILE *fp;

  fp = fopen(argv[1], "r");

  // first three values in file
  fscanf(fp, "%d", &numCycles);
  fscanf(fp, "%d", &width);
  fscanf(fp, "%d", &numInit);
  printf("# cycles %d width %d # initializations %d\n", numCycles, width, numInit);

//  printf("Tile size: %d\n", TILE_SIZE);
  u0 = (float *) calloc(width * width, sizeof(float));
  u1 = (float *) calloc(width * width, sizeof(float));

  // Create CUDA timers
  float elapsed_time;
  hipEvent_t start_timer, stop_timer;
  hipEventCreate(&start_timer);
  hipEventCreate(&stop_timer);

  // Allocate memory on device
  int memory_size = width * width * sizeof(float);
  float *device_u0, *device_u1;
  hipMalloc((void**) &device_u0, memory_size);
  hipMalloc((void**) &device_u1, memory_size);

  initGrid(u0, u1, width);

  for (n=0; n<numInit; n++) {
    fscanf(fp, "%d%d%lf", &i, &j, &inTemp);
    dataAt(u1, i, j, width) = inTemp;
  }

  // printGrid(u1, width);

  // Start timer
  hipEventRecord(start_timer, 0);

  // Copy from host to device
  hipMemcpy(device_u0, u0, memory_size, hipMemcpyHostToDevice);
  hipMemcpy(device_u1, u1, memory_size, hipMemcpyHostToDevice);

  // Tile size is limited by hardware, allowing a maximum of 1024 threads per block.
  // So max width for 2D grid is 32.
  int blockWidth = (width/10); // 10 is most optimal value
  int TILE_SIZE = width / blockWidth; // Round up

//  if (TILE_SIZE > 10)
//	  TILE_SIZE = 10;
  dim3 numBlocks ((width/TILE_SIZE), (width/TILE_SIZE));
  dim3 numThreads (TILE_SIZE, TILE_SIZE);

  printf("blockWidth: %d\n", ((width + TILE_SIZE -1)/TILE_SIZE));
  printf("width: %d, TILE_SIZE: %d\n", width, TILE_SIZE);
  printf("numBlocks: %d, numThreadsPerBlock: %d\n", (width*width)/(TILE_SIZE*TILE_SIZE), (TILE_SIZE*TILE_SIZE));

  for (cycle=0; cycle<numCycles; cycle++) {
    // updateGrid(u0, u1, width);
    // printGrid(u0, width);
    // Threads--

	/*
	 * Passing int args to kernel is only for 1D data sets. If you want to set up different data,
	 * have to declare different dims.
	 *
	 * Ex:
	 *
	 * 	dim2 numBlocks(width/TILE_SIZE, width/TILE_SIZE);
	 * 	dim2 numThreads(TILE_SIZE, TILE_SIZE);
	 *
	 * 	### ONLY WORKS FOR KERNEL CODE ###
	 * 	numBlocks.x = width/TILE_SIZE;
	 * 	numBlocks.y = width/TILE_SIZE;
	 *
	 * 	numThreads.x = TILE_SIZE;
	 * 	numThreads.y = TILE_SIZE;
	 *
	 *  ##################################
	 *
	 */



    updateGrid<<<numBlocks, numThreads>>>(device_u0, device_u1, width, TILE_SIZE);

    // Sync threads
    hipDeviceSynchronize();

    tptr = device_u0;
    device_u0 = device_u1;
    device_u1 = tptr;
  }

  // After finishing, copy data from device to host
  hipMemcpy(u0, device_u0, memory_size, hipMemcpyDeviceToHost);
  hipMemcpy(u1, device_u1, memory_size, hipMemcpyDeviceToHost);

  // Stop timer, calculate elapsed time
  hipEventRecord(stop_timer, 0);
  hipEventSynchronize(stop_timer);
  hipEventElapsedTime(&elapsed_time, start_timer, stop_timer);

  printf("Elapsed time = %f ms\n", elapsed_time);
  dumpGrid(u1, width);

  //printGrid(u1, width);

  // Free used vars
  free(u0);
  free(u1);
  hipFree(device_u0);
  hipFree(device_u1);

}
